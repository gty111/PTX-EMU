
#include <hip/hip_runtime.h>
#include<cstdio>
#define SIZE 1024
#define NUM 127

template<typename T>
__global__ void dummy_d(T *arr){
    arr[threadIdx.x] = threadIdx.x-NUM;
}

template<typename T>
bool dummy_h(){
    bool ifPASS = 0;
    T *a_h,*a_d;

    a_h = (T *)malloc(SIZE*sizeof(T));

    hipMalloc(&a_d,SIZE*sizeof(T));
    
    printf("host ptr:%p device ptr:%p\n",a_h,a_d);

    dummy_d<T> <<<1,SIZE>>>(a_d);

    hipMemcpy(a_h,a_d,SIZE*sizeof(T),hipMemcpyDeviceToHost);

    for(int i=0;i<SIZE;i++){
        if(a_h[i]!=i-NUM){
            printf("at:%p expect:%d got:%d\n",&a_h[i],i-NUM,a_h[i]);
            printf("FAIL\n");
            ifPASS = 1;
            goto End;
        }
    }
    printf("PASS\n");

    End:
    hipFree(a_d);
    free(a_h);

    return ifPASS;
}

int main(){
    return dummy_h<int>();
}   